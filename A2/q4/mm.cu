#include<iostream>
#include<hip/hip_runtime.h>
#include<stdio.h>
using namespace std;


__global__  void matrixmul(int* d_a,int* d_b,int* d_c,int r1,int c2,int r2)
{
   int row = blockIdx.y*blockDim.y + threadIdx.y;
   int col = blockIdx.x*blockDim.x + threadIdx.x;
   
if(row<r1 && col <c2)
   {
      int sum = 0;
      for(int i=0; i<r2; i++)
     {
            sum  = sum +  d_a[row*r2+i]*d_b[i*c2+col];
      }
     d_c[row*c2 + col] = sum;
   }
}


int main()
{
int r1=30,r2=30,c1=30,c2 = 30;
int *a= new int[r1*c1];
int *b= new int[r2*c2];
int *c= new int[r1*c2];
for(int i=0;i<r1;i++)
for(int j=0;j<c1;j++)

   { a[i*c1+j] = rand()%100;}

for(int i=0;i<r2;i++)
 { for(int j=0;j<c2;j++)
      { b[i*c2+j]= rand()%100;}}
if(r2!=c1)
{
cout<<"not possible";
return 0;
}
else
{    
int *d_a,*d_b,*d_c;
hipMalloc((void**)&d_a,sizeof(int)*r1*c1);
hipMalloc((void**)&d_b,sizeof(int)*r2*c2);
hipMalloc((void**)&d_c,sizeof(int)*r1*c2);
hipMemcpy(d_a,a,sizeof(int)*r1*c1,hipMemcpyHostToDevice);
hipMemcpy(d_b,b,sizeof(int)*r2*c2,hipMemcpyHostToDevice);
const dim3 blocksize(ceil(r1-1)/16 +1,ceil(c2-1)/16+1,1);
const dim3 gridsize(16,16,1);
//matrixmul<<<dim3(50,50),1>>>(d_a,d_b,d_c,r1,c2,r2);
matrixmul<<<blocksize,gridsize>>>(d_a,d_b,d_c,r1,c2,r2);

hipMemcpy(c,d_c,sizeof(int)*r1*c2,hipMemcpyDeviceToHost);

for(int i=0;i<r1;i++)
 { for(int j=0;j<c2;j++)
       { cout<<c[i*c2+j]<<" ";
}
 cout<<endl;}


hipFree(d_a);
hipFree(d_b);
hipFree(d_c);
}
return 0;


}
   



