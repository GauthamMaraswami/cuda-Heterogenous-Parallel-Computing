#include<iostream>
#include<hip/hip_runtime.h>
#include<stdio.h>
using namespace std;
#define TILE_WIDTH 4
__global__ void MatrixMulKernel( int* A, int* B, int* C,int m, int n, int k)
{
	__shared__ int ds_A[TILE_WIDTH][TILE_WIDTH];
	__shared__ int ds_B[TILE_WIDTH][TILE_WIDTH];
	int bx = blockIdx.x; int by = blockIdx.y;
	int tx = threadIdx.x; int ty = threadIdx.y;
	int cx = blockIdx.x * blockDim.x; int cy = blockIdx.y * blockDim.y;
	int Cx = cx + tx; int Cy = cy + ty;
	int Row = by * blockDim.y + ty;
	int Col = bx * blockDim.x + tx;
	int Cvalue = 0;
	int total_tiles = (n + TILE_WIDTH - 1) / TILE_WIDTH;
	for (int t = 0; t <  total_tiles; ++t)
	 {
		int Ax = t * TILE_WIDTH + tx; int Ay = cy + ty;
		int Bx = cx + tx; int By = t * TILE_WIDTH + ty;
		
		if (Ax < n && Ay < m) {
      			ds_A[ty][tx] = A[Ay * n + Ax];
   		 }
   		 else {
      			ds_A[ty][tx] = 0;
   		 }
    		if (Bx < n && By < k) {
     			ds_B[ty][tx] = B[By * n + Bx];
    		}
   		 else {
     			 ds_B[ty][tx] = 0;
		}
		__syncthreads();
		for (int i = 0; i < TILE_WIDTH; i++) {
      			Cvalue+= ds_A[ty][i] * ds_B[i][tx];
		}
		__syncthreads();
	}
		 if (Cx < n && Cy < m) {
    			C[Cy * n + Cx] = Cvalue;
  			}
}

int main()
{
int r1=30,r2=30,c1=30,c2 = 30;
int *a= new int[r1*c1];
int *b= new int[r2*c2];
int *c= new int[r1*c2];
for(int i=0;i<r1;i++)
for(int j=0;j<c1;j++)

   { a[i*c1+j] = rand()%100;}

for(int i=0;i<r2;i++)
 { for(int j=0;j<c2;j++)
      { b[i*c2+j]= rand()%100;}}
if(r2!=c1)
{
cout<<"not possible";
return 0;
}
else
{    
int *d_a,*d_b,*d_c;
hipMalloc((void**)&d_a,sizeof(int)*r1*c1);
hipMalloc((void**)&d_b,sizeof(int)*r2*c2);
hipMalloc((void**)&d_c,sizeof(int)*r1*c2);
hipMemcpy(d_a,a,sizeof(int)*r1*c1,hipMemcpyHostToDevice);
hipMemcpy(d_b,b,sizeof(int)*r2*c2,hipMemcpyHostToDevice);

const dim3 blocksize((c2/TILE_WIDTH) + 1, (r1/TILE_WIDTH) + 1, 1);//Number of Blocks required
const dim3 gridsize(TILE_WIDTH, TILE_WIDTH, 1);//Number of threads in each block


/*const dim3 blocksize(ceil(r1-1)/16 +1,ceil(c2-1)/16+1,1);
const dim3 gridsize(16,16,1);*/
//matrixmul<<<dim3(50,50),1>>>(d_a,d_b,d_c,r1,c2,r2);
MatrixMulKernel<<<blocksize,gridsize>>>(d_a,d_b,d_c,r1,c2,r2);

hipMemcpy(c,d_c,sizeof(int)*r1*c2,hipMemcpyDeviceToHost);

for(int i=0;i<r1;i++)
 { for(int j=0;j<c2;j++)
       { cout<<c[i*c2+j]<<" ";
}
 cout<<endl;}
 cout<<endl;

hipFree(d_a);
hipFree(d_b);
hipFree(d_c);
}
return 0;


}
   




