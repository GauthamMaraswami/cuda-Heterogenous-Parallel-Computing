#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include<stdio.h>
#include <stdlib.h>
#include<iostream>
int num;
float*  read_data(char c1[])
{
//printf("%s",c1);
FILE *file = fopen(c1, "r");
int i=0;
   // int num=0;
    fscanf(file, "%d", &num) ;
//        printf("%d",num);
  //      i++;
float * fobj=(float *)malloc(sizeof(float)*num);

float fint;
for( i=0;i<num;++i)
{
 fscanf(file, "%f", &fint) ;
	fobj[i]=fint;        
//printf("%f",fint);
       // i++;
}	
  
//for(int i=0;i<num;++i)
//printf("%f",fobj[i]);  
return fobj;



}
int main(int argc, char *argv[]) {

//  float *hostInput1 = nullptr;
 // float *hostInput2 = nullptr;
 // float *hostOutput = nullptr;
  
int inputLength;
int i;
std::cout<<num;
//return 0; 
float *input1=read_data(argv[3]);
float *input2=read_data(argv[2]);
float *expectedoutput=read_data(argv[1]);
/*for(int i=0;i<num;++i)
printf("%f",input1[i]);
*/ float *hostInput1 = input1;
float *hostInput2 = input2;
 float *hostOutput = NULL;

  /* parse the input arguments */

  //@@ Insert code here

  // Import host input data
thrust::host_vector<float> h_vec1(num);
for(int i=0;i<num;++i)
h_vec1[i]=input1[i];
thrust::host_vector<float> h_vec2(num);
for(int i=0;i<num;++i)
h_vec2[i]=input2[i];
thrust::host_vector<float> h_out(num);

/*for(int i = 0; i < num; i++)
 std::cout << "H[" << i << "] = " << h_vec1[i] << std::endl;
for(int i = 0; i < num; i++)
 std::cout << "H1[" << i << "] = " << h_vec2[i] << std::endl;*/
  //@@ Read data from the raw files here
  //@@ Insert code here
//  hostInput1 =
// hostInput2 =

  // Declare and allocate host output
  //@@ Insert code here
hostOutput=(float *)malloc(sizeof(float)*num);
  // Declare and allocate thrust device input and output vectors
  //@@ Insert code here

  // Copy to device
  //@@ Insert code here
thrust::device_vector<float> d_vec1 = h_vec1;
thrust::device_vector<float> d_vec2 = h_vec2;
thrust::device_vector<float>d_out(num);
  // Execute vector addition
  //@@ Insert Code here
thrust::transform(d_vec1.begin(),d_vec1.end(),d_vec2.begin(),d_out.begin(),thrust::plus<float>());
  /////////////////////////////////////////////////////////
h_out=d_out;

FILE *outp = fopen(argv[4], "w");
	fprintf(outp, "%d", num);
	for(int i = 0; i <num; ++i)
	{
	  fprintf(outp, "\n%.2f", h_out[i]);
	}






float *recievedoutput=read_data(argv[4]);
int matchflag=0;
for(long i = 0; i < 3987; i++)
 {
 // std::cout <<i<<"--"<<recievedoutput[i]<<"\t"; 
if(fabs(recievedoutput[i] - expectedoutput[i]) >= 0.001)
  {
	matchflag=i+1;
std::cout<<" xxxzzz"<<recievedoutput[i]<<"ccc"<<expectedoutput[i]<<"\n";
	break;
}
}
std::cout<<"xxxxxx"<<matchflag;
if(matchflag==0)
std::cout<<" matched";
else
 std::cout<<"not matched";
  // Copy data back to host
  //@@ Insert code here

  free(hostInput1);
  free(hostInput2);
  free(hostOutput);
  
return 0;
}

