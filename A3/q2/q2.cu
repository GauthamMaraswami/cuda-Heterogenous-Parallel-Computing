
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_WIDTH 256

__global__ void histogram(char *d_array_in, int *d_array_out, int n)
{
    __shared__ int shared_bin[128];

    int i, index, blocks, iterations;

    blocks = (n - 1) / BLOCK_WIDTH + 1; 
    iterations = 127 / (blocks * BLOCK_WIDTH) + 1;

    for (i = 0; i < iterations; i++) 
    {
        index = (blockIdx.x + i * blocks) * blockDim.x + threadIdx.x;

        if (index < 128)
        {
            d_array_out[index] = 0; 
        }
    }

    iterations = 127 / BLOCK_WIDTH + 1; 

    for (i = 0; i < iterations; i++)
    {
        index = i * blockDim.x + threadIdx.x;

        if (index < 128)
        {
            shared_bin[index] = 0; 
        }

        __syncthreads(); 
    }

    index = blockIdx.x * blockDim.x + threadIdx.x; 

    if (index < n)
    {
        atomicAdd(&shared_bin[d_array_in[index]], 1); 
    }

    __syncthreads();

    for (i = 0; i < iterations; i++) 
    {
        index = i * blockDim.x + threadIdx.x;

        if (index < 128)
        {
            atomicAdd(&d_array_out[index], shared_bin[index]); 
        }

        __syncthreads(); 
    }

    return;
}

int main(int argc, char *argv[])
{
    bool input_check = false;
    bool expected_check = false;
    bool output_check = false;
    bool error_present = false;
    bool expect_output = false;
    bool output_pass;

    char input_file_name[256];
    char expected_file_name[256];
    char output_file_name[256];

    FILE *input_file = NULL;
    FILE *expected_file = NULL;
    FILE *output_file = NULL;

    char *h_array_in = NULL;
    int *h_array_out = NULL;
    char *d_array_in = NULL;
    int *d_array_out = NULL;
    int *expectedOutput = NULL;

    int i, n, num_bins, dataset_no;

 

    for (i = 1; i < argc; i++)
    {
        if (strcmp(argv[i], "-i") == 0 && argc > i + 1)
        {
            if (argv[i + 1][0] != '-')
            {
                input_check = true;

                strcpy(input_file_name, argv[i + 1]);
            }
        }

        if (strcmp(argv[i], "-e") == 0 && argc > i + 1)
        {
            if (argv[i + 1][0] != '-')
            {
                expected_check = true;

                strcpy(expected_file_name, argv[i + 1]);
            }
        }

        if (strcmp(argv[i], "-o") == 0)
        {
            expect_output = true;

            if (argc > i + 1)
            {
                if (argv[i + 1][0] != '-')
                {
                    output_check = true;

                    strcpy(output_file_name, argv[i + 1]);
                }
            }
        }
    }

    if (!input_check)
    {
        std::cout << "Execution command syntax error: \"Input\" filename required" << std::endl;

        error_present = true;
    }
    else
    {
        input_file = fopen(input_file_name, "r");

        if (!input_file)
        {
            std::cout << "Error: File " << input_file_name << " does not exist" << std::endl;

            error_present = true;
        }
    }

    if (!expected_check)
    {
        std::cout << "Execution command syntax error: \"Expected Output\" filename required" << std::endl;

        error_present = true;
    }
    else
    {
        expected_file = fopen(expected_file_name, "r");

        if (!expected_file)
        {
            std::cout << "Error: File " << expected_file_name << " does not exist" << std::endl;

            error_present = true;
        }
    }

    if (!output_check && expect_output)
    {
        std::cout << "Execution Command Syntax Warning: \"Output\" filename expected" << std::endl;
    }
    else if (output_check)
    {
        output_file = fopen(output_file_name, "w");
    }

    if (error_present)
    {
        std::cout << "Use the following command to run the program:\n\n"
                     "./<program> ­-e <expected> -­i <input> ­-o <output>\n\n"
                     "Where <expected> is the expected output file, <input> is the input dataset files, and <output> is an optional path to store the results"
                  << std::endl;
    }
    else
    {
        dataset_no = 0;

        while (true)
        {
          
            h_array_in = (char *)malloc(1024 * sizeof(char));

            if (fgets(h_array_in, 1024, input_file) == NULL)
            {
                break;
            }

            for (n = 0; h_array_in[n] != '\n'; n++)
            {
                continue;
            }

            h_array_in[n] = '\0';

            if (fscanf(expected_file, "%d", &num_bins) == -1)
            {
                break;
            }

            expectedOutput = (int *)malloc(num_bins * sizeof(int));

            for (i = 0; i < num_bins; i++)
            {
                fscanf(expected_file, "%d", &expectedOutput[i]);
            }

            h_array_out = (int *)malloc(128 * sizeof(int));

            
            hipMalloc((void **)&d_array_in, n * sizeof(char));
            hipMalloc((void **)&d_array_out, 128 * sizeof(int));

            
            hipMemcpy(d_array_in, h_array_in, n * sizeof(char), hipMemcpyHostToDevice);

            
            dim3 blocks((n - 1) / BLOCK_WIDTH + 1);
            dim3 threads_per_block(BLOCK_WIDTH); 

            
            histogram<<<blocks, threads_per_block>>>(d_array_in, d_array_out, n);

           
            hipMemcpy(h_array_out, d_array_out, 128 * sizeof(int), hipMemcpyDeviceToHost);

            
            if (output_check)
            {
                fprintf(output_file, "%d", 128);

                for (i = 0; i < num_bins; i++)
                {
                    fprintf(output_file, "\n%d", h_array_out[i]);
                }

                fprintf(output_file, "\n");

                fflush(output_file);
            }

           
            output_pass = true;
            for (i = 0; i < 128; i++)
            {
                if (expectedOutput[i] != h_array_out[i])
                {
                    output_pass = false;
                }
            }

            if (output_pass)
            {
                std::cout << "Dataset " << dataset_no << " PASSED" << std::endl;
            }
            else
            {
                std::cout << "Dataset " << dataset_no << " FAILED" << std::endl;
            }

            dataset_no++;

          
            hipFree(d_array_in);
            hipFree(d_array_out);

            free(h_array_in);
            free(h_array_out);
            free(expectedOutput);
        }

        if (output_check)
        {
            std::cout << "Results stored in " << output_file_name << std::endl;
        }

        fclose(input_file);
        fclose(expected_file);

        if (output_check)
        {
            fclose(output_file);
        }
    }

    return 0;
}
