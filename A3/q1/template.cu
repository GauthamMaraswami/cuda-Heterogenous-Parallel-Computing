#include "hip/hip_runtime.h"
#include "wb.h"
#include <iostream>
#include <hip/hip_runtime.h>
using namespace std;
#define NUM_BINS 4096

#define CUDA_CHECK(ans)                                                   \
  { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code),
            file, line);
    if (abort)
      exit(code);
  }
}

__global__ void hist_kernel(unsigned int *deviceInput, unsigned int *deviceBins,unsigned int inputLength)
{
  int i = threadIdx.x + blockDim.x*blockIdx.x;
  
 if(i<inputLength)
    {int item = deviceInput[i];  atomicAdd(&(deviceBins[item]),1);}
 
}

__global__ void bins_cap(unsigned int *deviceBins)
{
  int i = threadIdx.x + blockDim.x*blockIdx.x;
  deviceBins[i] = min(127,deviceBins[i]);
}

int main(int argc, char *argv[]) {

 unsigned  int inputLength;
  unsigned int *hostInput;
  unsigned int *hostBins;
  unsigned int *deviceInput;
  unsigned int *deviceBins;

 
  wbArg_t args = wbArg_read(argc,argv);
  FILE* inp = fopen(argv[1],"r");
	fscanf(inp,"%d",&inputLength);

  wbTime_start(Generic, "Importing data and creating memory on host");
	
  hostInput = new unsigned int[inputLength];
  
  for(int i=0;i<inputLength;i++)
  {
    fscanf(inp,"%d",&hostInput[i]);
  }
  
  hostBins = (unsigned int *)malloc(NUM_BINS * sizeof(int));
  
  wbTime_stop(Generic, "Importing data and creating memory on host");

  wbLog(TRACE, "The input length is ", inputLength);
  wbLog(TRACE, "The number of bins is ", NUM_BINS);

  wbTime_start(GPU, "Allocating GPU memory.");
  //@@ Allocate GPU memory here
  cout<<inputLength<<endl;
  int size = inputLength*sizeof(int);

  hipMalloc((void **)&deviceInput,size);
  hipMalloc((void **)&deviceBins,NUM_BINS*sizeof(unsigned int));
  hipMemset(deviceBins,0,NUM_BINS*sizeof(unsigned int));
  CUDA_CHECK(hipDeviceSynchronize());

  wbTime_stop(GPU, "Allocating GPU memory.");
  wbTime_start(GPU, "Copying input memory to the GPU.");

  //@@ Copy memory to the GPU here
  hipMemcpy(deviceInput,hostInput,size,hipMemcpyHostToDevice);
  
  CUDA_CHECK(hipDeviceSynchronize());
  wbTime_stop(GPU, "Copying input memory to the GPU.");

  // Launch kernel
  // ----------------------------------------------------------
  wbLog(TRACE, "Launching kernel");
  wbTime_start(Compute, "Performing CUDA computation");
  //@@ Perform kernel computation here

  int threads = 1024;
  hist_kernel<<<(inputLength-1)/1024+1, threads>>>(deviceInput,deviceBins,inputLength);
  bins_cap<<<4,1024>>>(deviceBins);
  wbTime_stop(Compute, "Performing CUDA computation");

  wbTime_start(Copy, "Copying output memory to the CPU");
  //@@ Copy the GPU memory back to the CPU here
  hipMemcpy(hostBins,deviceBins,NUM_BINS*sizeof(unsigned int),hipMemcpyDeviceToHost);
  CUDA_CHECK(hipDeviceSynchronize());
/*  
for(int i=0;i<NUM_BINS;i++)
  {
    cout<<hostBins[i]<<endl;
  }*/
  wbTime_stop(Copy, "Copying output memory to the CPU");

  wbTime_start(GPU, "Freeing GPU Memory");
  //@@ Free the GPU memory here
  hipFree(deviceInput);
  hipFree(deviceBins);
  wbTime_stop(GPU, "Freeing GPU Memory");


/*cout<<"\nHello \n";
for(int i=0;i<1024;i++){
  //cout<<hostBins[i]<<endl;
//hostBins[i]=127;
}
cout<<"\n\nDone\n";*/
  // Verify correctness
  // -----------------------------------------------------
  wbSolution(args, hostBins, NUM_BINS);
  free(hostBins);
  free(hostInput);
  return 0;
}
