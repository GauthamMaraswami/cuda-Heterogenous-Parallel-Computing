//Submitted by GAutham M 15co118 and yashwanth 15co154
#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include <time.h>
__global__ void func(float *da_in,float *db_in,float *d_out)
{
  int idx = blockIdx.x*100 + threadIdx.x;
   d_out[idx] = da_in[idx] + db_in[idx];
}

int main()
{
const int array_size = 16000;
const int array_bytes = array_size* sizeof(float);
float a_in[array_size],b_in[array_size];
for(int i=0;i<array_size;i++)
{
   a_in[i] = float(i);
   b_in[i]=float(i);
}
/*for(int i=0;i<array_size;i++)
{
   b_in[i]=rand()%16000;
}*/
float h_out[array_size];
float *da_in;
float *db_in;
float *d_out;
int temp=array_size;
int array_bytes1=array_bytes;
time_t t,t1;
srand((unsigned) time(&t));
 t1=clock();
while(temp>1)
{
//printf("abc");
if((temp)%2==1)
{
a_in[temp]=0;
//printf("con fail\n");
temp++;
array_bytes1+=8;
}
temp=temp/2;

array_bytes1/=2;
hipMalloc((void **)&da_in,array_bytes1);
hipMalloc((void **)&db_in,array_bytes1);
hipMalloc((void **)&d_out,array_bytes1);

hipMemcpy(da_in,a_in,array_bytes1,hipMemcpyHostToDevice);
hipMemcpy(db_in,a_in+(temp),array_bytes1,hipMemcpyHostToDevice);
//kernel
func<<<dim3(160,1,1),dim3(100,1,1)>>>(da_in,db_in,d_out);

//copying back
hipMemcpy(h_out,d_out,array_bytes1,hipMemcpyDeviceToHost);

for(int i=0;i<temp;i++)
{
//      a_in[i]=h_out[i];
  // printf("%d=%f",i+1,h_out[i]);
  // printf(((i%4)!=3)? "\t":"\n");
  a_in[i]=h_out[i];
}
hipFree(da_in);
hipFree(d_out);
hipFree(db_in);
//printf("\n");

}

 t1=clock()-t1;
double time_taken = ((double)t1)/CLOCKS_PER_SEC;
printf("parallel execution gave answer as%f- time taken as %f\n",a_in[0],time_taken);
}