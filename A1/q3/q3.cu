//15co154 Yeshwanth R
//15co118 Goutham M


#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>

__global__ void func(float *da_in,float *db_in,float *d_out)
{
  int idx = blockIdx.x*100 + threadIdx.x;
   d_out[idx] = da_in[idx] + db_in[idx];
}





int main()
{
float t1,t2;
const int array_size = 16000;
const int array_bytes = array_size* sizeof(float);
float a_in[array_size],b_in[array_size];

for(int i=0;i<array_size;i++)
{
   a_in[i] = float(i);
}
for(int i=0;i<array_size;i++)
{
   b_in[i]=rand()%16000;
}
float h_out[array_size];
float *da_in;
float *db_in;
float *d_out;
hipMalloc((void **)&da_in,array_bytes);
hipMalloc((void **)&db_in,array_bytes);
hipMalloc((void **)&d_out,array_bytes);

hipMemcpy(da_in,a_in,array_bytes,hipMemcpyHostToDevice);
hipMemcpy(db_in,b_in,array_bytes,hipMemcpyHostToDevice);
//kernel
func<<<dim3(160,1,1),dim3(100,1,1)>>>(da_in,db_in,d_out);
float time;
//copying back
hipMemcpy(h_out,d_out,array_bytes,hipMemcpyDeviceToHost);
for(int i=0;i<array_size;i++)
{
   printf("%f",h_out[i]);
   printf(((i%12)!=3)? "\t":"\n");
}
hipFree(da_in);
hipFree(d_out);
hipFree(db_in);
printf("\n\n\n\n");


}
