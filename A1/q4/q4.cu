//15co154 Yeshwanth R
//15co118 Goutham M
#include<stdio.h>
#include<hip/hip_runtime.h>

__global__ void addition(int *da_in ,int *db_in ,int *d_out){


     int idx = blockIdx.x*5 + threadIdx.x;
     int idy = blockIdx.y*5 + threadIdx.y;
      int in = idx + idy*5;
        d_out[in] = da_in[in] + db_in[in];

}

int main()
{
int array_size = 10;
int array_bytes = 10*10*sizeof(int);
int a_in[array_size][array_size],b_in[array_size][array_size];
int h_out[array_size][array_size];
for(int i=0;i<array_size;i++)
for(int j=0;j<array_size;j++)
    a_in[i][j] = i;

for(int i=0;i<array_size;i++)
for(int j=0;j<array_size;j++)
    b_in[i][j] = j;

int *da_in;
int *db_in;
int *d_out;

hipMalloc((void **)&da_in,array_bytes);
hipMalloc((void **)&db_in,array_bytes);
hipMalloc((void **)&d_out,array_bytes);


hipMemcpy(da_in,a_in,array_bytes,hipMemcpyHostToDevice);
hipMemcpy(db_in,b_in,array_bytes,hipMemcpyHostToDevice);

addition<<<dim3(2,2,1),dim3(5,5,1)>>>(da_in,db_in,d_out);


hipMemcpy(h_out,d_out,array_bytes,hipMemcpyDeviceToHost);

for(int i=0;i<array_size;i++)
{
for(int j=0;j<array_size;j++)

{
   printf("%d  ",h_out[i*10+j]);
  }
printf("\n");
}
hipFree(da_in);
hipFree(db_in);
hipFree(d_out);
}