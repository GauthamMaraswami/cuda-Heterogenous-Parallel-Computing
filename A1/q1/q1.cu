//Submitted by GAutham M 15co118 and yashwanth 15co154
#include <stdio.h>

int main() {
  int nDevices;

  hipGetDeviceCount(&nDevices);
  for (int i = 0; i < nDevices; i++) {
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		printf("Device Number: %d\n", i);
		printf("  Device name: %s\n", prop.name);
		printf("  Memory Clock Rate (KHz): %d\n",
			   prop.memoryClockRate);
		printf("  Memory Bus Width (bits): %d\n",
			   prop.memoryBusWidth);
		printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
			   2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
		printf("Total global memory:           %u\n",  prop.totalGlobalMem);
		printf("Total shared memory per block: %u\n",  prop.sharedMemPerBlock);
		printf("Total registers per block:     %d\n",  prop.regsPerBlock);
		printf("Warp size:                     %d\n",  prop.warpSize);
		printf("Maximum memory pitch:          %u\n",  prop.memPitch);
		printf("Maximum threads per block:     %d\n",  prop.maxThreadsPerBlock);
		for (int i = 0; i < 3; ++i)
	    printf("Maximum dimension %d of block:  %d\n", i, prop.maxThreadsDim[i]);
		for (int i = 0; i < 3; ++i)
		printf("Maximum dimension %d of grid:   %d\n", i, prop.maxGridSize[i]);
		printf("Total constant memory:         %u\n",  prop.totalConstMem);
		printf("Texture alignment:             %u\n",  prop.textureAlignment);
		printf("Concurrent copy and execution: %s\n",  (prop.deviceOverlap ? "Yes" : "No"));
		printf("Number of multiprocessors:     %d\n",  prop.multiProcessorCount);
		printf("Kernel execution timeout:      %s\n",  (prop.kernelExecTimeoutEnabled ? "Yes" : "No"));


}

}